#include "hip/hip_runtime.h"
﻿#include "LatinSquares.cuh"
#include "Utils.h"
#include "MIN.h"
#include <math.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>



#define OUTFILE1    "./Results/results-ls.txt"
#define OUTFILE2    "./Results/results-mols.txt"
#define DEBUG_LOG   "./Results/log.txt"

#define CUDA_ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "%s: %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void cuda_handle_error() {
	auto err = hipGetLastError();
	printf("%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
}

void cuda_main() {

	const unsigned int N = 16;
	const unsigned int SWITCHES = N / 2;
	const unsigned int STAGES = (2 * log2(N)) - 1;

	const unsigned int MOLS_GRID_PARALLEL = 10;
	const unsigned int MOLS_SEQ_CUTOFF = 8;
	const unsigned int NUM_LATIN_SQUARES = MOLS_GRID_PARALLEL * MOLS_SEQ_CUTOFF;
	// const unsigned int BLOCK = 10;
	
	const unsigned int LS_SAMPLES = 200;
	const bool DO_COMPLETE_CHECK = true;
	const bool DEBUG = true;

	freopen(DEBUG_LOG, "w+", stdout);

	/// COMPUTE LATIN SQUARES

	printf("Computing latin squares...\n");
	auto start_ls = clock();

	int* topology = new int[16 * 6];
	make_butterfly_butterfly_topology(topology);

	bool* char_mat = new bool[16 * 8 * 7];
	// make_characteristic_matrices(0, char_mat);
	generate_rotation_configurations(topology, char_mat);

	bool* dev_char_mat;
	int* dev_topology;
	bool* dev_conf;
	bool* dev_is_latin_square;
	int* dev_perm;
	hiprandState* dev_states1;

	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_char_mat, (16 * 8 * 7) * sizeof(bool)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_topology, (16 * 6) * sizeof(int)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_conf, (NUM_LATIN_SQUARES) * (8 * 7) * sizeof(bool)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_is_latin_square, (NUM_LATIN_SQUARES) * sizeof(bool)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_perm, (NUM_LATIN_SQUARES) * (16 * 16) * sizeof(int)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_states1, NUM_LATIN_SQUARES * sizeof(hiprandState)));

	CUDA_ERROR_CHECK(hipMemcpy(dev_char_mat, char_mat, (16 * 8 * 7) * sizeof(bool), hipMemcpyHostToDevice));
	CUDA_ERROR_CHECK(hipMemcpy(dev_topology, topology, (16 * 6) * sizeof(int), hipMemcpyHostToDevice));

	setup_rand_state << <NUM_LATIN_SQUARES, 1 >> > (dev_states1);
	cuda_handle_error();

	check_latin_square << <NUM_LATIN_SQUARES, 1 >> > (dev_states1, dev_char_mat, dev_topology, dev_conf, dev_is_latin_square, dev_perm);
	cuda_handle_error();

	delete[] topology;
	delete[] char_mat;
	CUDA_ERROR_CHECK(hipFree(dev_char_mat));
	CUDA_ERROR_CHECK(hipFree(dev_topology));
	CUDA_ERROR_CHECK(hipFree(dev_states1));

	bool* out_is_ls = new bool[NUM_LATIN_SQUARES];
	bool* out_conf = new bool[(NUM_LATIN_SQUARES) * (8 * 7)];
	int* out_perm = new int[(NUM_LATIN_SQUARES) * (16 * 16)];

	CUDA_ERROR_CHECK(hipMemcpy(out_is_ls, dev_is_latin_square, NUM_LATIN_SQUARES * sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipMemcpy(out_conf, dev_conf, (NUM_LATIN_SQUARES) * (8 * 7) * sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipMemcpy(out_perm, dev_perm, (NUM_LATIN_SQUARES) * (16 * 16) * sizeof(int), hipMemcpyDeviceToHost));

	CUDA_ERROR_CHECK(hipFree(dev_conf));

	FILE* fd_ls = fopen(OUTFILE1, "w+");
	write_output_latin_square(fd_ls, out_is_ls, out_conf, out_perm, NUM_LATIN_SQUARES);
	fclose(fd_ls);

	delete[] out_conf;
	delete[] out_is_ls;

	auto end_ls = clock();
	printf("Done in %6.4f ms.\n", (double)(end_ls - start_ls) / CLOCKS_PER_SEC);

	/// END: COMPUTE LATIN SQUARES


	/// COMPUTE MUTUALLY ORTHOGONAL LATIN SQUARES

	printf("Computing MOLS, complete check: %d...\n", DO_COMPLETE_CHECK);
	auto start_mols = clock();

	bool* dev_mols;
	int* dev_pairs;
	hiprandState* dev_states2;
	dim3 grid_size;
	unsigned int NUM_COMPARISONS;

	if (DO_COMPLETE_CHECK) {
		NUM_COMPARISONS = NUM_LATIN_SQUARES * NUM_LATIN_SQUARES;
		CUDA_ERROR_CHECK(hipMalloc((void**)&dev_mols, (NUM_COMPARISONS) * sizeof(bool)));
		CUDA_ERROR_CHECK(hipMalloc((void**)&dev_pairs, (NUM_COMPARISONS * 2) * sizeof(int)));
		grid_size = dim3(NUM_LATIN_SQUARES, MOLS_GRID_PARALLEL, 1);
		check_mols_complete << < grid_size, MOLS_SEQ_CUTOFF >> > (dev_perm, dev_is_latin_square, dev_mols, dev_pairs, DEBUG);
		cuda_handle_error();
		CUDA_ERROR_CHECK(hipDeviceSynchronize());
	}
	else {
		NUM_COMPARISONS = NUM_LATIN_SQUARES * LS_SAMPLES;
		CUDA_ERROR_CHECK(hipMalloc((void**)&dev_states2, (NUM_COMPARISONS) * sizeof(hiprandState)));
		CUDA_ERROR_CHECK(hipMalloc((void**)&dev_mols, (NUM_COMPARISONS) * sizeof(bool)));
		CUDA_ERROR_CHECK(hipMalloc((void**)&dev_pairs, (NUM_COMPARISONS * 2) * sizeof(int)));
		setup_rand_state << <NUM_LATIN_SQUARES, LS_SAMPLES >> > (dev_states2);
		cuda_handle_error();
		check_mols_random << < NUM_LATIN_SQUARES, LS_SAMPLES >> > (dev_states2, dev_perm, dev_is_latin_square, dev_mols, dev_pairs, DEBUG);
		cuda_handle_error();
		CUDA_ERROR_CHECK(hipFree(dev_states2));
	}

	CUDA_ERROR_CHECK(hipFree(dev_is_latin_square));
	CUDA_ERROR_CHECK(hipFree(dev_perm));

	bool* out_mols = new bool[NUM_COMPARISONS];
	int* out_pairs = new int[NUM_COMPARISONS * 2];

	CUDA_ERROR_CHECK(hipMemcpy(out_mols, dev_mols, NUM_COMPARISONS * sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipMemcpy(out_pairs, dev_pairs, NUM_COMPARISONS * 2 * sizeof(int), hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipFree(dev_mols));
	CUDA_ERROR_CHECK(hipFree(dev_pairs));

	FILE* fd_mols = fopen(OUTFILE2, "w+");
	write_output_mols(fd_mols, out_mols, out_perm, out_pairs, NUM_COMPARISONS);
	fclose(fd_mols);

	auto end_mols = clock();
	printf("Done in %6.4f ms.\n", (double)(end_mols - start_mols) / CLOCKS_PER_SEC);



	/// END: COMPUTE MUTUALLY ORTHOGONAL LATIN SQUARES

	// release memory

	delete[] out_perm;
	delete[] out_mols;
	delete[] out_pairs;

}
