#include "hip/hip_runtime.h"
﻿#include "LatinSquares.cuh"
#include "Utils.h"
#include "MIN.h"
#include <math.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>



#define OUTFILE1 "./Results/results-ls.txt"
#define OUTFILE2 "./Results/results-mols.txt"

#define CUDA_ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "%s: %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void cuda_handle_error() {
	auto err = hipGetLastError();
	printf("%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
}

void cuda_main() {

	const int GRID = 10;
	const int BLOCK = 10;
	const int LS_SAMPLES = 5;

	/// COMPUTE LATIN SQUARES

	printf("Computing latin squares...\n");
	auto start_ls = clock();

	int* topology = new int[16 * 6];
	make_butterfly_butterfly_topology(topology);

	bool* char_mat = new bool[16 * 8 * 7];
	make_characteristic_matrices(0, char_mat);

	bool* dev_char_mat;
	int* dev_topology;
	bool* dev_conf;
	bool* dev_is_latin_square;
	int* dev_perm;
	hiprandState* dev_states1;

	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_char_mat, (16 * 8 * 7) * sizeof(bool)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_topology, (16 * 6) * sizeof(int)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_conf, (GRID * BLOCK) * (8 * 7) * sizeof(bool)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_is_latin_square, (GRID * BLOCK) * sizeof(bool)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_perm, (GRID * BLOCK) * (16 * 16) * sizeof(int)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_states1, GRID * BLOCK * sizeof(hiprandState)));

	CUDA_ERROR_CHECK(hipMemcpy(dev_char_mat, char_mat, (16 * 8 * 7) * sizeof(bool), hipMemcpyHostToDevice));
	CUDA_ERROR_CHECK(hipMemcpy(dev_topology, topology, (16 * 6) * sizeof(int), hipMemcpyHostToDevice));

	setup_rand_state << <GRID, BLOCK >> > (dev_states1);
	cuda_handle_error();

	check_latin_square << <GRID, BLOCK >> > (dev_states1, dev_char_mat, dev_topology, dev_conf, dev_is_latin_square, dev_perm);
	cuda_handle_error();

	delete[] topology;
	delete[] char_mat;
	CUDA_ERROR_CHECK(hipFree(dev_char_mat));
	CUDA_ERROR_CHECK(hipFree(dev_topology));
	CUDA_ERROR_CHECK(hipFree(dev_states1));

	bool* out_is_ls = new bool[GRID * BLOCK];
	bool* out_conf = new bool[(GRID * BLOCK) * (8 * 7)];
	int* out_perm = new int[(GRID * BLOCK) * (16 * 16)];

	CUDA_ERROR_CHECK(hipMemcpy(out_is_ls, dev_is_latin_square, GRID * BLOCK * sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipMemcpy(out_conf, dev_conf, (GRID * BLOCK) * (8 * 7) * sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipMemcpy(out_perm, dev_perm, (GRID * BLOCK) * (16 * 16) * sizeof(int), hipMemcpyDeviceToHost));

	CUDA_ERROR_CHECK(hipFree(dev_conf));

	FILE* fd_ls = fopen(OUTFILE1, "w+");
	write_output_latin_square(fd_ls, out_is_ls, out_conf, out_perm, GRID, BLOCK);
	fclose(fd_ls);

	delete[] out_conf;

	printf("Done in %6.4f ms.\n", clock() - start_ls);

	/// END: COMPUTE LATIN SQUARES


	/// COMPUTE MUTUALLY ORTHOGONAL LATIN SQUARES

	printf("Computing MOLS...\n");
	auto start_mols = clock();

	bool* dev_mols;
	int* dev_pairs;
	hiprandState* dev_states2;
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_states2, (GRID * BLOCK * LS_SAMPLES) * sizeof(hiprandState)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_mols, (GRID * BLOCK * LS_SAMPLES) * sizeof(bool)));
	CUDA_ERROR_CHECK(hipMalloc((void**)&dev_pairs, (GRID * BLOCK * LS_SAMPLES * 2) * sizeof(int)));

	auto grid_size = dim3(GRID, BLOCK, 1);

	setup_rand_state << <grid_size, LS_SAMPLES >> > (dev_states2);
	cuda_handle_error();

	check_mols << < grid_size, LS_SAMPLES >> > (dev_states2, dev_perm, dev_is_latin_square, dev_mols, dev_pairs);
	cuda_handle_error();

	bool* out_mols = new bool[GRID * BLOCK * LS_SAMPLES];
	int* out_pairs = new int[GRID * BLOCK * LS_SAMPLES * 2];

	CUDA_ERROR_CHECK(hipMemcpy(out_mols, dev_mols, GRID * BLOCK * LS_SAMPLES * sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipMemcpy(out_pairs, dev_pairs, GRID * BLOCK * LS_SAMPLES * 2 * sizeof(int), hipMemcpyDeviceToHost));

	FILE* fd_mols = fopen(OUTFILE2, "w+");
	write_output_mols(fd_mols, out_mols, out_perm, out_pairs, GRID, BLOCK, LS_SAMPLES);
	fclose(fd_mols);

	printf("Done in %6.4f ms.\n", clock() - start_mols);

	/// END: COMPUTE MUTUALLY ORTHOGONAL LATIN SQUARES

	// release memory

	CUDA_ERROR_CHECK(hipFree(dev_is_latin_square));
	CUDA_ERROR_CHECK(hipFree(dev_perm));
	CUDA_ERROR_CHECK(hipFree(dev_mols));
	CUDA_ERROR_CHECK(hipFree(dev_pairs));
	CUDA_ERROR_CHECK(hipFree(dev_states2));

	delete[] out_is_ls;
	delete[] out_perm;
	delete[] out_mols;
	delete[] out_pairs;

}
