#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include ""
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void helloCUDA(float f) {
    printf("hello thread %d, arg = %f", threadIdx.x, f);
}

int main() {
    helloCUDA << <1, 5 >> > (0.4f);
    hipDeviceSynchronize();
    return 0;
}